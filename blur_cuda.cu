#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

__global__ void blurKernel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    for (int c = 0; c < channels; ++c) {
        int pixVal = 0;
        int pixels = 0;

        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                int nx = x + dx;
                int ny = y + dy;

                if (nx >= 0 && ny >= 0 && nx < width && ny < height) {
                    pixVal += input[(ny * width + nx) * channels + c];
                    pixels++;
                }
            }
        }

        output[(y * width + x) * channels + c] = pixVal / pixels;
    }
}

int main() {
    cv::Mat input = cv::imread("input.jpg", cv::IMREAD_COLOR);
    if (input.empty()) {
        std::cerr << "Image not found!\n";
        return -1;
    }

    int imgSize = input.rows * input.cols * input.channels();

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);

    hipMemcpy(d_input, input.data, imgSize, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((input.cols + 15) / 16, (input.rows + 15) / 16);

    blurKernel<<<grid, block>>>(d_input, d_output, input.cols, input.rows, input.channels());

    cv::Mat output(input.size(), input.type());
    hipMemcpy(output.data, d_output, imgSize, hipMemcpyDeviceToHost);

    cv::imwrite("output_cuda.jpg", output);

    hipFree(d_input);
    hipFree(d_output);

    std::cout << "CUDA blur completed.\n";
    return 0;
}
